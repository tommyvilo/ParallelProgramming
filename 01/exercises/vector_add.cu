
#include <hip/hip_runtime.h>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>

const int N = 1 << 22; //shif a sinistra di 22, 2^22 -> 4194304

using timer = std::chrono::high_resolution_clock;

#define cuda_check_error()                                                     \
  {                                                                            \
    hipError_t e = hipGetLastError();                                        \
    if (e != hipSuccess) {                                                    \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__,                 \
             hipGetErrorString(e));                                           \
      std::exit(EXIT_FAILURE);                                                 \
    }                                                                          \
  }

#define cuda_safe_call(x)                                                      \
  {                                                                            \
    x;                                                                         \
    cuda_check_error();                                                        \
  }

//@@ Insert code to implement vector addition
__global__ void mainKernel(int* d_a,int* d_b,int* d_o,int N) {

  int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if(globalIndex<N) //evito di scrivere in aree di memoria no allocate (parlo per i threads di troppo che ho)
    d_o[globalIndex] = d_a[globalIndex] + d_b[globalIndex];

}

int main() {

  // HOST memory
  int *h_a = new int[N];
  int *h_b = new int[N];
  int *h_o = new int[N];

  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);
  std::uniform_int_distribution<int> distribution(1, 100);

  for (int i = 0; i < N; i++) {
    h_a[i] = distribution(generator);
    h_b[i] = distribution(generator);
  }

  int *h_o_real = new int[N]; // Used to check the result
  auto t0 = timer::now();

  for (int i = 0; i < N; i++) {
    h_o_real[i] = h_a[i] + h_b[i];
  }

  auto t1 = timer::now();
  auto host_duration = t1 - t0;
  std::cout << "HOST version time: "
            << std::chrono::duration_cast<std::chrono::microseconds>(
                   host_duration)
                   .count()
            << " us" << std::endl;

  // -------------------------------------------------------------------------
  // Exercise

  int *d_a, *d_b, *d_o; // <- Device memory pointers

  //@@ Allocate GPU memory
  cuda_safe_call(hipMalloc(&d_a, N*sizeof(int)));
  cuda_safe_call(hipMalloc(&d_b, N*sizeof(int)));
  cuda_safe_call(hipMalloc(&d_o, N*sizeof(int)));

  //@@ Copy memory to the GPU
  cuda_safe_call(hipMemcpy(d_a,h_a,N*sizeof(int),hipMemcpyHostToDevice));
  cuda_safe_call(hipMemcpy(d_b,h_b,N*sizeof(int),hipMemcpyHostToDevice));

  //@@ Initialize the grid and block dimensions
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock; 

  t0 = timer::now();

  //@@ Launch the GPU Kernel
  cuda_safe_call((mainKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_o, N)));
  hipDeviceSynchronize();

  t1 = timer::now();
  auto gpu_duration = t1 - t0;

  std::cout << "GPU version time: "
            << std::chrono::duration_cast<std::chrono::microseconds>(
                   gpu_duration)
                   .count()
            << " us" << std::endl;

  std::cout << std::setprecision(1)
            << "speedup: " << host_duration / gpu_duration << "x\n\n";

  //@@ Copy the GPU memory back to the CPU here
  cuda_safe_call(hipMemcpy(h_o,d_o,N*sizeof(int),hipMemcpyDeviceToHost));

  //@@ Free the GPU memory here
  cuda_safe_call(hipFree(d_a));
  cuda_safe_call(hipFree(d_b));
  cuda_safe_call(hipFree(d_o));


  // -------------------------------------------------------------------------
  // RESULT CHECK

  for (int i = 0; i < N; i++) {
    if (h_o[i] != h_o_real[i]) {
      std::cerr << "wrong result at: " << i << "\nhost:   " << h_o_real[i]
                << "\ndevice: " << h_o[i] << "\n\n";
      return 1;
    }
  }
  std::cout << "<> Correct\n\n";

  delete[] h_a;
  delete[] h_b;
  delete[] h_o;

  return 0;
}
